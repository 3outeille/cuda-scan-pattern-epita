#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"
#include "cuda_tools/host_shared_ptr.cuh"

#include <hip/hip_runtime_api.h>


template <typename T>
__global__
void kernel_scan_baseline(T* buffer, int size)
{
    for (int i = 1; i < size; ++i)
        buffer[i] += buffer[i - 1];
}

void baseline_scan(cuda_tools::host_shared_ptr<int> buffer)
{
    hipProfilerStart();
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_scan_baseline<int>), hipFuncCachePreferShared);

	kernel_scan_baseline<int><<<1, 1>>>(buffer.data_, buffer.size_);

    hipDeviceSynchronize();
    kernel_check_error();
    
    hipProfilerStop();
}

template <typename T>
__global__
void kernel_your_scan(T* buffer, int size)
{
    // TODO
    // ...
}

void your_scan(cuda_tools::host_shared_ptr<int> buffer)
{
    hipProfilerStart();

    // TODO
    // ...

	kernel_your_scan<int><<<1, 1>>>(buffer.data_, buffer.size_);

    hipDeviceSynchronize();
    kernel_check_error();
    
    hipProfilerStop();
}