#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"
#include "cuda_tools/host_shared_ptr.cuh"

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <limits>

template <typename T>
static __global__ void single_pass_chained_scan(T *buffer, T *reduce_arr, int buffer_size, int reduce_arr_size)
{
    int tid = threadIdx.x;
    int id = threadIdx.x + (blockIdx.x * blockDim.x);

    __shared__ int shared_memory[1024];

    if (id < buffer_size)
        shared_memory[tid] = buffer[id];

    __syncthreads();

    // Reduce
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if (tid % (2 * stride) == 0)
            shared_memory[tid] += shared_memory[tid + stride];
        __syncthreads();
    }

    if (tid == 0)
        reduce_arr[blockIdx.x] = shared_memory[0];

    __syncthreads();

    // Adjacent synchronization
    if (tid == 0 && blockIdx.x > 0) {
        // Wait until all previous block finish to compute its reduce.
        for (int i = blockIdx.x - 1; i >= 0; i--)
        {
            // Force reduce_arr to be in global memory (and not register)
            while (atomicAdd(&reduce_arr[i], 0) == std::numeric_limits<T>::min())
            {
            }
        }

        // Add associate reduce_arr to 1st element of each block of buffer.
        for (int i = blockIdx.x - 1; i >= 0; i--)
            buffer[blockIdx.x * blockDim.x] += reduce_arr[i];
    }

    __syncthreads();

    // Scan
    int x;

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if (tid >= stride && (id % blockDim.x) != 0)
            x = buffer[id] + buffer[id - stride];

        __syncthreads();

        if (tid >= stride && (id % blockDim.x) != 0)
            buffer[id] = x;

        __syncthreads();
    }
}

void scan_opti_4_slow(cuda_tools::host_shared_ptr<int> buffer)
{
    // Single-pass chained-scan using reduce-then-scan
    // TODO: seems to be veryslow
    hipProfilerStart();

    constexpr int nb_threads = 1024;
    const int nb_blocks = (buffer.size_ + nb_threads - 1) / nb_threads;

    int *tmp;
    hipMalloc(&tmp, nb_blocks * sizeof(int));
    hipMemset(tmp, std::numeric_limits<int>::min(), nb_blocks * sizeof(int));

    single_pass_chained_scan<int><<<nb_blocks, nb_threads>>>(buffer.data_, tmp, buffer.size_, nb_blocks);
    hipDeviceSynchronize();
    kernel_check_error();

    hipProfilerStop();
}