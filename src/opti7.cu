#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"
#include "cuda_tools/host_shared_ptr.cuh"

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <limits>

template <typename T, int BLOCK_SIZE>
// Single pass chained scan Scan-then-propagate
static __global__ void decoupled_look_back_stp(T* input, T* flag, T* record_local_sum, T* record_total_cum_sum, int* block_id,
                                                    int input_size) {
    int tid = threadIdx.x;
    __shared__ int bid;
    
    // id of each blocks are now based on shcedulation of block and not by blockIdx.x
    // This is to avoid deadlock when block 0 cannot be scheduled on GPU because GPU is full
    if (tid == 0)
        bid = atomicAdd(block_id, 1);
    
    __syncthreads();

    int gid = threadIdx.x + (bid * blockDim.x);

    __shared__ int shared_memory[BLOCK_SIZE];

    if (gid < input_size)
        shared_memory[tid] = input[gid];
    else
        shared_memory[tid] = 0; // To avoid undefined behavior

    __syncthreads();

    // Scan
    #pragma unroll
    for (int step = 0; (1 << step) < BLOCK_SIZE; step += 1)
    {
        int left = 1 << step;
        if ((tid & left) != 0) {
            int right = tid >> step;
            int from = left * right - 1;
            shared_memory[tid] += shared_memory[from];
        }

        __syncthreads();
    }

    // Decoupled look-back (adjacent synchronization)
    __shared__ int value_to_propagate;
    
    if (tid == 0) {
        value_to_propagate = 0;

        if (bid == 0) {
            // First block to be scheduled on SM must have the total sum (since it cannot look-back)
            atomicAdd(&record_total_cum_sum[0], shared_memory[BLOCK_SIZE - 1]);
            atomicExch(&flag[0], 2); // Set to P
        } else {
            // Block has computed its local reduce
            atomicAdd(&record_local_sum[bid], shared_memory[BLOCK_SIZE - 1]);
            atomicExch(&flag[bid], 1); // set to A

            int prev_bid = bid - 1;

            while (true) {
                int prev_flag;
                // Look at previous immediate block
                do {
                    prev_flag = atomicAdd(&flag[prev_bid], 0);
                } while (prev_flag == 0);

                if (prev_flag == 2) {
                    // The block has the total sum, we can stop here
                    int prev_record_total_cum_sum = atomicAdd(&record_total_cum_sum[prev_bid], 0);
                    value_to_propagate += prev_record_total_cum_sum;
                    atomicAdd(&record_total_cum_sum[bid], value_to_propagate + shared_memory[BLOCK_SIZE - 1]);
                    atomicExch(&flag[bid], 2); // Set to P
                    break;
                } else {
                    // The block has only its local sum, we need to look-back
                    int prev_record_local_sum = atomicAdd(&record_local_sum[prev_bid], 0);
                    value_to_propagate += prev_record_local_sum;
                    prev_bid -= 1;
                    continue;
                }
            }
        }
    }

    __syncthreads();

    if (gid < input_size)
        input[gid] = shared_memory[tid] + value_to_propagate;
}

void scan_opti_7(cuda_tools::host_shared_ptr<int> buffer) {
    // Decoupled look back using scan_then_propagate
    // constexpr int nb_threads = 1024;
    constexpr int nb_threads = 1024;
    const int nb_blocks = (buffer.size_ + nb_threads - 1) / nb_threads;

    // 0: X / 1: A / 2: P
    int* flag;
    int* record_local_sum;
    int* record_total_cum_sum;
    int* block_id;

    hipMalloc(&flag, nb_blocks * sizeof(int));
    hipMalloc(&record_local_sum, nb_blocks * sizeof(int));
    hipMalloc(&record_total_cum_sum, nb_blocks * sizeof(int));
    hipMalloc(&block_id, sizeof(int));

    hipMemset(flag, 0, nb_blocks * sizeof(int));
    hipMemset(record_local_sum, 0, nb_blocks * sizeof(int));
    hipMemset(record_total_cum_sum, 0, nb_blocks * sizeof(int));
    hipMemset(block_id, 0, sizeof(int));

    hipProfilerStart();

    // Single pass chained scan Scan-then-propagate
    decoupled_look_back_stp<int, nb_threads>
        <<<nb_blocks, nb_threads>>>(buffer.data_, flag, record_local_sum, record_total_cum_sum, block_id, buffer.size_);
    hipDeviceSynchronize();
    kernel_check_error();

    hipProfilerStop();
}